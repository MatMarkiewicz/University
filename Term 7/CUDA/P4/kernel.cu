#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

#include "hip/hip_runtime_api.h"

////////////////////////////////////////////////////////////////////////////////
// CPU routines
////////////////////////////////////////////////////////////////////////////////

void reduction_gold(float* odata, float* idata, const unsigned int len)
{
    *odata = 0;
    for (int i = 0; i < len; i++) *odata += idata[i];
}

////////////////////////////////////////////////////////////////////////////////
// GPU routines
////////////////////////////////////////////////////////////////////////////////

__global__ void reduction(float* g_odata, float* g_idata)
{
    // dynamically allocated shared memory

    extern  __shared__  float temp[];

    int tid = threadIdx.x;
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    // first, each thread loads data into shared memory

    temp[id] = g_idata[id];

    // V1 (zad 3 i 4)

    // find previout power of 2
    // int k = 1;
    // int bd = blockDim.x;
    // while (bd >>= 1)
    //     k = k << 1;

    // __syncthreads();
    // if (tid < blockDim.x - k) temp[id] += temp[id + k];

    // next, we perform binary tree reduction

    // for (int d = k >> 1; d > 0; d >>= 1) {
    //     __syncthreads();  // ensure previous step completed 
    //     if (tid < d)  temp[id] += temp[id + d];
    // }

    // finally, first thread puts result into global memory

 
    // if (tid == 0) g_odata[blockIdx.x] = temp[blockDim.x * blockIdx.x];


    // V2 (zad 5)

    int value = temp[id];
    int bd = blockDim.x;
    for (int i = bd/2; i > 0; i = i / 2)
        value += __shfl_down_sync(-1, value, i);
    if (tid == 0) g_odata[blockIdx.x] = value;
}





////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main(int argc, const char** argv)
{
    int num_elements, num_threads, mem_size, shared_mem_size, num_blocks;

    float* h_data, * reference, sum, gpu_sum;
    float* d_idata, * d_odata;

    // initialise card

    findCudaDevice(argc, argv);

    num_elements = 512;
    num_blocks = 16;
    num_threads = num_elements/num_blocks;
    mem_size = sizeof(float) * num_elements;

    // allocate host memory to store the input data
    // and initialize to integer values between 0 and 1000

    h_data = (float*)malloc(mem_size);

    for (int i = 0; i < num_elements; i++)
        h_data[i] = floorf(1000 * (rand() / (float)RAND_MAX));

    // compute reference solutions

    reference = (float*)malloc(mem_size);
    reduction_gold(&sum, h_data, num_elements);

    // allocate device memory input and output arrays

    checkCudaErrors(hipMalloc((void**)&d_idata, mem_size));
    checkCudaErrors(hipMalloc((void**)&d_odata, sizeof(float)* num_blocks));

    // copy host memory to device input array

    checkCudaErrors(hipMemcpy(d_idata, h_data, mem_size,
        hipMemcpyHostToDevice));

    // execute the kernel

    shared_mem_size = sizeof(float) * num_elements;
    reduction << <num_blocks, num_threads, shared_mem_size >> > (d_odata, d_idata);
    getLastCudaError("reduction kernel execution failed");

    // copy result from device to host

    checkCudaErrors(hipMemcpy(h_data, d_odata, sizeof(float)* num_blocks,
        hipMemcpyDeviceToHost));

    gpu_sum = 0.0;
    for (int k = 0; k < num_blocks; k++) gpu_sum += h_data[k];

    // check results

    printf("reduction error = %f\n", gpu_sum - sum);

    // cleanup memory

    free(h_data);
    free(reference);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    // CUDA exit -- needed to flush printf write buffer

    hipDeviceReset();
}