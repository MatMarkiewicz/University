// Zad5:
// (2 * 4B * 100 (pętla) * 960000 (NPATH)) / 1.5ms
// 512GB/s (COLAB - TESLA T4)
// kod prezentowany na pracowni, działa z printem w kernelu, bez printa zwrca nan

////////////////////////////////////////////////////////////////////////
// GPU version of Ax2+bz+c
////////////////////////////////////////////////////////////////////////

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>


////////////////////////////////////////////////////////////////////////
// CUDA global constants
////////////////////////////////////////////////////////////////////////

__constant__ int a,b,c;

////////////////////////////////////////////////////////////////////////
// kernel routine
////////////////////////////////////////////////////////////////////////

__global__ void average(float *d_rands, float *d_res) {
    int ind = threadIdx.x + blockIdx.x*blockDim.x;
    //printf("ID %d %d %d %d\n", ind, threadIdx.x, blockIdx.x, blockDim.x);
    float sum, z;

    for (int n=0; n<100; n++) {
        z = d_rands[ind];
        sum += a*z*z + b*z + c;
        ind += 32;
    }
    
    d_res[threadIdx.x + blockIdx.x*blockDim.x] = sum / 100.0f;
}

int main(int argc, const char **argv){
    int N=3200;
    int A=5;
    int B=1;
    int C=12;
    float *h_res, *d_res, *d_rands;

    h_res = (float *)malloc(N/100*sizeof(float));
    hipMalloc((void **)&d_res, N/100*sizeof(float));
    hipMalloc((void **)&d_rands, N*sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(a), &A, sizeof(A));
    hipMemcpyToSymbol(HIP_SYMBOL(b), &B, sizeof(B));
    hipMemcpyToSymbol(HIP_SYMBOL(c), &C, sizeof(C));

    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    hiprandGenerateNormal(gen, d_rands, N, 0.0f, 1.0f);

    average<<<2, 16>>>(d_rands, d_res);
    hipDeviceSynchronize();
    hipMemcpy(h_res, d_res, sizeof(float)*N/100, hipMemcpyDeviceToHost);
    float res;
    for (int i=0;i<N/100;i++){
        printf("h: %f\n", h_res[i]);
        res += h_res[i];
    }
    printf("Result: %f\n", res/(N/100));

    hiprandDestroyGenerator(gen);
    free(h_res);
    hipFree(d_res);
    hipFree(d_rands);

    hipDeviceReset();
}