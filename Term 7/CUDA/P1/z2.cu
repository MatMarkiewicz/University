#include "hip/hip_runtime.h"
// zadanie prezentowane na pracowni

//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>

__managed__  float x[16];

//
// kernel routine
// 

__global__ void my_first_kernel()
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  x[tid] = (float) threadIdx.x;
}


//
// main code
//

int main(int argc, const char **argv)
{
  int   nblocks, nthreads, nsize, n; 

  // initialise card

  findCudaDevice(argc, argv);

  // set number of blocks, and threads per block

  nblocks  = 2;
  nthreads = 8;
  nsize    = nblocks*nthreads ;

  // allocate memory for array

  // execute kernel
  
  my_first_kernel<<<nblocks,nthreads>>>();
  getLastCudaError("my_first_kernel execution failed\n");

  // synchronize to wait for kernel to finish, and data copied back

  hipDeviceSynchronize();

  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %f \n",n,x[n]);

  // free memory 

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

  return 0;
}