// zadanie prezentowane na pracowni

//
// include files
//


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

//
// kernel routine
// 

__global__ void my_first_kernel(float *x, float *y, float *z)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  z[tid] = x[tid] + y[tid];
}


//
// main code
//

int main(int argc, char **argv)
{
  float *h1_x, *h2_x, *h3_x, *d1_x, *d2_x, *d3_x;
  int   nblocks, nthreads, nsize, n; 

  // set number of blocks, and threads per block

  nblocks  = 2;
  nthreads = 8;
  nsize    = nblocks*nthreads ;

  // allocate memory for array

  h1_x = (float *)malloc(nsize*sizeof(float));
  h2_x = (float *)malloc(nsize*sizeof(float));
  h3_x = (float *)malloc(nsize*sizeof(float));
  hipMalloc((void **)&d1_x, nsize*sizeof(float));
  hipMalloc((void **)&d2_x, nsize*sizeof(float));
  hipMalloc((void **)&d3_x, nsize*sizeof(float));
 
  for (int i=0;i<nsize;i++){
      h1_x[i] = 0.11*i;
      h2_x[i] = 0.33*i;
  }

  hipMemcpy(d1_x,h1_x,nsize*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d2_x,h2_x,nsize*sizeof(float),hipMemcpyHostToDevice);

  // execute kernel

  my_first_kernel<<<nblocks,nthreads>>>(d1_x, d2_x, d3_x);

  // copy back results and print them out

  hipMemcpy(h3_x,d3_x,nsize*sizeof(float),hipMemcpyDeviceToHost);

  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %f \n",n,h3_x[n]);

  // free memory 

  hipFree(d1_x);
  hipFree(d2_x);
  hipFree(d3_x);
  free(h1_x);
  free(h2_x);
  free(h3_x);

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

  return 0;
}